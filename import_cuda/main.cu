#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipcub/hipcub.hpp"
#include <cstdio>


template <int BLOCK_SIZE>
__global__ void reduce_sum(float *d_x, float *d_y, int count) {
  int g_id = threadIdx.x + blockDim.x * blockIdx.x;
  using BlockRudce = hipcub::BlockReduce<float, BLOCK_SIZE>;
  __shared__ typename BlockRudce::TempStorage t_sum;
  float x = g_id < count ? d_x[g_id] : 0.0f;
  __syncthreads();
  float sum = BlockRudce(t_sum).Reduce(x, hipcub::Sum());
  if (threadIdx.x == 0)
    atomicAdd(d_y, sum);
}

float ReduceSum(float *x, int count, hipStream_t stream = 0) {
    float sum = 0.0f;
    float *h_x, *d_x_and_y;
    hipError_t status = hipHostMalloc(&h_x, sizeof(float) * count);

    if (status != hipSuccess) {
      printf("fail to malloc host\n");
      return sum;
    }
    

    status = hipMalloc(&d_x_and_y, sizeof(float) * (count + 1));
    if (status != hipSuccess) {
      hipFree(h_x);
      printf("fail to malloc device\n");
      return sum;
    }

    for (int i = 0; i < count; ++i) {
        h_x[i] = x[i];
    }

    const int nBlock = (count + 383) / 384;
    status = hipMemcpyAsync(d_x_and_y, h_x, count * sizeof(float), hipMemcpyHostToDevice, stream);
    if (status != hipSuccess) {
      printf("fail to copy to device\n");
      goto error_break;
    }
    
    reduce_sum<384><<<nBlock, 384, 0, stream>>>(d_x_and_y, d_x_and_y + count, count);
    status = hipMemcpyAsync(h_x, d_x_and_y + count, sizeof(float), hipMemcpyDeviceToHost, stream);
        if (status != hipSuccess) {
        
      printf("fail to copy to host\n");
      goto error_break;
    }

    status = hipStreamSynchronize(stream);
    if (status != hipSuccess) {
        
      printf("fail to sync\n");
      goto error_break;
    }

    
    sum = h_x[0];


    error_break:
      hipFree(h_x);
      hipFree(d_x_and_y);
      
      return sum;
    

}

int main() {

    float *x = new float[384];
    for (int i = 0; i < 384; ++i) {
        x[i] = i * 0.01f;
    }

    hipStream_t stream;
    hipStreamCreate(&stream);
    float sum = ReduceSum(x, 384, stream);

    float true_sum = (383) * (384 / 2) * 0.01f;

    hipStreamDestroy(stream);
    printf("%f %f\n", sum, true_sum);

    delete x;
    return 0;
}